//Basic Hellow World with cuda


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void)
{
    printf("Hello World from GPU!\n");
}


int main(void)
{
    printf("Hello World from CPU!\n");

    helloFromGPU<<<1, 10>>>(); //launch 10 blocks of 1 thread each

    hipDeviceReset();//wait for GPU to finish before exiting
    return 0;
}
