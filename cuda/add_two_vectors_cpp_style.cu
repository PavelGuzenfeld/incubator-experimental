#include <algorithm>
#include <array>
#include <hip/hip_runtime.h>
#include <stdio.h>

constexpr int N = 512;
constexpr int MAX_BLOCK_SIZE = 1024; // Typical max block size

__global__ void add(int *a, int *b, int *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

class GpuArray
{
public:
    explicit GpuArray(size_t n) : size_(n * sizeof(int))
    {
        hipMalloc(&data_, size_);
    }

    ~GpuArray()
    {
        hipFree(data_);
    }

    void copyToDevice(const std::array<int, N> &host_array)
    {
        hipMemcpy(data_, host_array.data(), size_, hipMemcpyHostToDevice);
    }

    void copyToHost(std::array<int, N> &host_array)
    {
        hipMemcpy(host_array.data(), data_, size_, hipMemcpyDeviceToHost);
    }

    int *data() const
    {
        return data_;
    }

private:
    int *data_ = nullptr;
    size_t size_;
};

int main()
{
    std::array<int, N> a, b, c;

    // fill with random numbers
    for (auto &val : a)
    {
        val = rand() % 100;
    }
    for (auto &val : b)
    {
        val = rand() % 100;
    }

    GpuArray d_a(N), d_b(N), d_c(N);

    d_a.copyToDevice(a);
    d_b.copyToDevice(b);

    constexpr int block_size = (N < MAX_BLOCK_SIZE) ? N : MAX_BLOCK_SIZE;
    constexpr int grid_size = (N + block_size - 1) / block_size;
    add<<<grid_size, block_size>>>(d_a.data(), d_b.data(), d_c.data());

    // check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch kernel: %s\n", hipGetErrorString(err));
        return 1;
    }

    d_c.copyToHost(c);

    // print results
    for (int i = 0; i < N; i++)
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    return 0;
}
