// Add two vectors on the GPU

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define N 512

__global__ void add(int *a, int *b, int *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

void random_ints(int *a, int n)
{
    for (int i = 0; i < n; i++)
        a[i] = rand() % 100; // or any other range you prefer
}

int main(void) {
    int *a, *b, *c; // host copies of a, b, c
    int *d_a, *d_b, *d_c; // device copies of a, b, c
    int size = N * sizeof(int);

    // Alloc space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Alloc space for host copies of a, b, c and setup input values
    a = (int *)malloc(size); random_ints(a, N);
    b = (int *)malloc(size); random_ints(b, N);
    c = (int *)malloc(size);

    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch add() kernel on GPU with better block and thread configuration
    add<<<(N + 127) / 128, 128>>>(d_a, d_b, d_c);

    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch kernel: %s\n", hipGetErrorString(err));
        // handle error...
    }

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Print results
    for (int i = 0; i < N; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // Cleanup
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}

